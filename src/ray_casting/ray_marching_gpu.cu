#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "ray_marching/ray_marching_gpu.hpp"

__global__ void cuda_ray_marching(Particle_t* particles,
                                  float* distMap,
                                  float* rays_angle,
                                  int n_particles,
                                  Cloud_t* cloud,
                                  Map_t* map,
                                  float* rays)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int nCast = n_particles * N_RAYS_DS;
    if (idx >= nCast)
        return;
    int pIDX = (float)idx / float(N_RAYS_DS);
    int aIDX = fmodf(idx, N_RAYS_DS); // idx % angleToCast;

    // int iix = int(aIDX * angleDownsample);
    float angle =
      (particles[pIDX].yaw + cloud->angleMin +
       rays_angle[aIDX]); // angleMin) + (float(aIDX) * float(angleDownsample)) * angleIncrement;

    // generate ray with rayMarching
    float rayPoseX = particles[pIDX].x;
    float rayPoseY = particles[pIDX].y;

    float t = 0.0f;
    float out = cloud->maxRange;
    while (t < cloud->maxRayIteration) {
        int c = (int)((map->opp_originX - rayPoseX) / map->map_resolution);
        int r = (int)((map->opp_originY + rayPoseY) / map->map_resolution);

        if (c < 0 || c >= map->map_width || r < 0 || r > map->map_height) {
            out = cloud->maxRange;
            break;
        }
        int i = r * map->map_width + c;
        float distance = distMap[i];
        rayPoseX += distance * std::cos(angle);
        rayPoseY += distance * std::sin(angle);

        if (distance <= map->map_resolution) {
            float xd = rayPoseX - particles[pIDX].x;
            float yd = rayPoseY - particles[pIDX].y;
            out = sqrtf(xd * xd + yd * yd);
            break;
        }

        t += fmaxf(distance * 0.999f, 1.0);
    }
    rays[idx] = out;
}

// turn this into a constructor, move hipMemcpyAsync?
void RayMarchingGPU::init(Particle_t* particles,
                        float* distMap,
                        Cloud_t* cloud,
                        Map_t* map,
                        int n_particles,
                        float* rays_angle)
{
    hipStreamCreate(&stream);

    hipMalloc((void**)&d_distGrid, sizeof(float) * map->map_height * map->map_width);
    hipMalloc((void**)&d_particles, sizeof(Particle_t) * n_particles);
    hipMalloc((void**)&d_cloud, sizeof(Cloud_t));
    hipMalloc((void**)&d_map, sizeof(Map_t));
    hipMalloc((void**)&d_raysAngle, sizeof(float) * N_RAYS_DS);
    hipMalloc((void**)&d_rays, sizeof(float) * n_particles * N_RAYS_DS);

    hipMemcpyAsync(d_cloud, cloud, sizeof(Cloud_t), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_map, map, sizeof(Map_t), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_distGrid,
                    distMap,
                    sizeof(float) * map->map_height * map->map_width,
                    hipMemcpyHostToDevice,
                    stream);
    hipMemcpyAsync(
      d_raysAngle, &rays_angle[0], sizeof(float) * N_RAYS_DS, hipMemcpyHostToDevice, stream);
}

void RayMarchingGPU::castRays(Particle_t* particles, int n_particles, float* rays)
{
    // copy to gpu
    hipMemcpyAsync(
      d_particles, &particles[0], sizeof(Particle_t) * n_particles, hipMemcpyHostToDevice, stream);

    int nCast = n_particles * N_RAYS_DS;
    int blockSize = 192;
    int numBlocks = (nCast + blockSize - 1) / blockSize;

    cuda_ray_marching<<<numBlocks, blockSize, 0, stream>>>(
      d_particles, d_distGrid, d_raysAngle, n_particles, d_cloud, d_map, d_rays);

    hipMemcpyAsync(
      &rays[0], d_rays, sizeof(float) * n_particles * N_RAYS_DS, hipMemcpyDeviceToHost, stream);
    hipStreamSynchronize(stream);
}

void RayMarchingGPU::close()
{
    hipFree(d_distGrid);
    hipFree(d_particles);
    hipFree(d_cloud);
    hipFree(d_map);
}